#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    const int row = blockDim.y * blockIdx.y + threadIdx.y;
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row >=rows || col >= cols) {
        return;
    }

    const int inputIdx = row * cols + col;
    const int outputIdx = col * rows + row;

    output[outputIdx] = input[inputIdx];

}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}